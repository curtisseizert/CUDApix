#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include "sieve/phisieve_device.cuh"
#include "sieve/phisieve_host.cuh"

void Phisieve::init(uint16_t c)
{
  sieveCountInit<<<blocks, threads>>>(d_sieve, d_count, bstart, c);
  hipDeviceSynchronize();

  thrust::inclusive_scan(thrust::device, d_count, d_count + blockSize, d_count);
  hipDeviceSynchronize();
  
  countFinit<<<blocks, threads>>>(d_count);
}
