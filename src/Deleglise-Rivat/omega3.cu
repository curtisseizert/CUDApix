#include "hip/hip_runtime.h"
// omega3.cu
//
// 128-bit
//
// A segmented implemenation of the sum "C" in Xavier Gourdon's variant of the
// Deleglise-Rivat prime counting algorithm with modified bounds to simplify
// implementation with a range of y values.
//
// Copywrite (c) 2016 Curtis Seizert <cseizert@gmail.com>

#include <iostream>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_uint128.h>
#include <CUDASieve/cudasieve.hpp>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <CUDASieve/host.hpp>
#include <math_functions.h>
#include <hip/hip_runtime_api.h>

#include "general/tools.hpp"
#include "general/device_functions.cuh"
#include "Deleglise-Rivat/A.cuh"
#include "Deleglise-Rivat/deleglise-rivat.hpp"
#include "cudapix.hpp"
#include "pitable.cuh"

uint128_t deleglise_rivat128::omega3()
{
  hipStream_t stream[3];
  uint64_t sum = 0;
  PrimeArray pq(0, y);
  uint64_t sqrty = _isqrt(y);
  uint64_t pi_sqrty = CudaSieve::countPrimes(0, sqrty);

  uint64_t num_p = pi_qrtx - pi_sqrty;

  pq.d_primes = CudaSieve::getDevicePrimes(pq.bottom, pq.top, pq.len, 0);

  uint64_t maxblocks = 1 + ((pq.len - pi_qrtx)/threadsPerBlock);

  uint64_t * d_sums;
  uint64_t * d_lastQ, * d_nextQ;
  hipMalloc(&d_sums, maxblocks * sizeof(uint64_t));
  hipMalloc(&d_lastQ, num_p * sizeof(uint64_t));
  hipMalloc(&d_nextQ, num_p * sizeof(uint64_t));
  hipHostMalloc(&pq.h_primes, pq.len * sizeof(uint64_t));
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);
  hipStreamCreate(&stream[2]);

  hipMemcpy(pq.h_primes, pq.d_primes, pq.len*sizeof(uint64_t), hipMemcpyDeviceToHost);
  global::Omega3_lower_bound<<<num_p/threadsPerBlock + 1, threadsPerBlock, 0, stream[1]>>>
    (x, lastQ, pq.d_primes, pi_sqrty, pi_qrtx);
  global::zero<<<maxblocks, threadsPerBlock, 0, stream[2]>>>(d_sums, maxblocks);
  thrust::upper_bound(thrust::device, lastQ, lastQ + num_p, pq.d_primes, pq.d_primes + pq.len, lastQ);

  hipDeviceSynchronize();
  KernelTime timer;
  timer.start();

  PiTable pi_table(sqrtx, z/qrtx);
  pi_table.set_pi_base(pi_sqrtx);

  // find all (p,q) pairs such that x/(p * q) >= x^(3/8)
  while(pi_table.get_base() > pq.top){
    // nextQ is copied from lastQ each iteration rather than switching pointers
    // to form the basis of a compare and swap operation in the kernel that evaluates
    // whether a given value has changed
    hipMemcpyAsync(d_nextQ, d_lastQ, num_p * sizeof(uint64_t), hipMemcpyDeviceToDevice, stream[1]);

    // get this iteration's pi table and bounds
    uint64_t pi_max = pi_table.get_pi_base();
    uint32_t * d_piTable = pi_table.getNextDown();

    // launch kernel
    hipDeviceSynchronize();
    hipProfilerStart();
    Omega3_kernel<<<maxblocks, threadsPerBlock, 0, stream[0]>>>
      (x, y, pq.d_primes, d_piTable, (pi_table.get_pi_base() & (~1ull)), pi_table.get_base(),
      pMaxIdx, d_sums, d_nextQ, d_lastQ, (uint64_t)pq.len);
    hipProfilerStop();
    hipDeviceSynchronize();

  }

  sum = thrust::reduce(thrust::device, d_sums, d_sums + maxblocks);
  std::cout << "Omega 3:\t" << sum << std::endl;

  timer.stop();
  timer.displayTime();
  timer.~KernelTime();
  pi_table.~PiTable();

  hipFree(d_sums);
  hipFree(pq.d_primes);
  hipHostFree(pq.h_primes);

  hipDeviceReset();

  return sum + sum2;
}

__global__
void Omega3_kernel( uint128_t x, uint64_t y, uint64_t * pq, uint32_t * d_pitable,
                    uint64_t pi_0, uint64_t base, uint32_t pMaxIdx, uint64_t * sums,
                    uint64_t * nextQ, uint64_t * lastQ, uint64_t maxQidx)
{
  uint64_t sum = 0;
  uint32_t tidx = threadIdx.x + blockDim.x * blockIdx.x;
  __shared__ uint64_t s_pi[numThreads];
  s_pi_chi[threadIdx.x] = 0;
  __shared__ uint64_t s_lastQ[numThreads];

  for(uint64_t j = 0; j < pMaxIdx - 1; j += numThreads){
    s_lastQ[threadIdx.x] = (uint64_t)-1;
    __syncthreads();
    for(uint64_t i = j; i < min((uint32_t)pMaxIdx, (uint32_t)j + numThreads); i++){

      uint64_t qidx = nextQ[i] + tidx;
      if(qidx >= maxQidx){
        atomicCAS((unsigned long long *)&s_lastQ[0], (unsigned long long)-1, (unsigned long long)maxQidx);
        break;
      }
      uint64_t q = pq[qidx];
      uint64_t p = pq[i];

      // calculate x/(p * q) and store value in q
      q = uint128_t::div128(x, (p * q));

      // check to make sure quotient is > pi_0, and coordinate this block's value
      // of lastQ if not
      q = checkRange(q, base, s_lastQ[i % numThreads], qidx);

      // calculate pi(x/(p * q)) * chi(x/(p * q)) if q is in range
      if(q != 0)
        s_pi_chi[threadIdx.x] += calculatePiChi(q, y, d_pitable, pi_0, base);
    } // repeat for all p values in range
    __syncthreads();

    // get global minimum value of lastQ
    minLastQ(j, s_lastQ, nextQ, lastQ);
  }
  __syncthreads();
  sum = thrust::reduce(thrust::device, s_pi_chi, s_pi_chi + numThreads);
  if(threadIdx.x == 0)
    sums[blockIdx.x] += sum;
}

__device__
inline uint64_t checkRange(uint64_t q, uint64_t base, uint64_t & s_lastQ, uint64_t qidx)
{
  if(q + 1 < base){
    atomicMin((unsigned long long *)&s_lastQ, (unsigned long long)qidx);
    q = 0;
  }
  return q;
}

__device__
inline uint64_t calculatePiChi(uint64_t q, uint64_t y, uint32_t * d_pitable,
                                uint64_t pi_0, uint64_t base)
{
  // uint64_t r = d_pitable[(q + 1 - (base & ~1ull))/2] + pi_0;

  // for some reason doing this with ptx cuts about 5% off overall run time
  uint64_t r;
  uint32_t *ptr = &d_pitable[(q + 1 - (base & ~1ull))/2];
  asm("ld.global.u32.ca   %0, [%1];\n\t"
       : "=l" (r)
       : "l" (ptr));
  r += pi_0;

  if(q < y)
    r <<= 1;
  return r;
}

__device__
inline void minLastQ(uint32_t j, uint64_t * s_lastQ, uint64_t * nextQ, uint64_t * lastQ)
{
  uint32_t i = j + threadIdx.x;
  if(s_lastQ[threadIdx.x] != ~0){
    atomicCAS((unsigned long long *)&lastQ[i], (unsigned long long)nextQ[i], (unsigned long long)s_lastQ[threadIdx.x]);
    atomicMin((unsigned long long *)&lastQ[i], (unsigned long long)s_lastQ[threadIdx.x]);
  }
}

__global__
void Omega3_lower_bound(uint128_t x, uint64_t * nextQ, uint64_t * pq,
                        uint64_t p0Idx, uint64_t pMaxIdx)
{
  uint64_t tidx = threadIdx.x + blockDim.x * blockIdx.x;
  uint64_t pidx = p0Idx + tidx;
  uint64_t p = pq[pidx];

  if(pidx <= pMaxIdx)
    nextQ[tidx] = x / (p * p * p)
}
