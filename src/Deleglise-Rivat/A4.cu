#include "hip/hip_runtime.h"
// A4.cu
//
// 128-bit
//
// A segmented implemenation of the sum "A" in Xavier Gourdon's variant of the
// Deleglise-Rivat prime counting algorithm with an upper bound less constrained
// by the memory requirements of holding pi(x) values from 0 to sqrt(x) as in
// A1.cu
//
// Copywrite (c) 2016 Curtis Seizert <cseizert@gmail.com>

#include <iostream>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_uint128.h>
#include <CUDASieve/cudasieve.hpp>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <CUDASieve/host.hpp>
#include <math_functions.h>
#include <hip/hip_runtime_api.h>

#include "general/tools.hpp"
#include "general/device_functions.cuh"
#include "Deleglise-Rivat/A.cuh"
#include "Deleglise-Rivat/deleglise-rivat.hpp"
#include "cudapix.hpp"
#include "pitable.cuh"

uint128_t deleglise_rivat128::A()
{
// this is a segmented variant of the above, where each iteration evaluates
// all of the pi(x/(p * q)) values that fall within the range of the pi table.
// The basic idea is that each time the maximum value of q is exceeded (as defined
// by the minimum of x/(p*q) imposed by the minimum value of our pi table), the
// id of this element of the array will be saved in a different array, which will
// serve as the first q used for each p in the next iteration (with the next smaller
// pi table)
  hipStream_t stream[3];
  uint64_t sum = 0;
  PrimeArray pq(qrtx, pow(sqrtx, 0.75));

  uint64_t num_p = pi_cbrtx - pi_qrtx;

  pq.d_primes = CudaSieve::getDevicePrimes(pq.bottom, pq.top, pq.len, 0);

  uint64_t maxblocks = 1 + (pq.len/threadsPerBlock);

  uint64_t * d_sums;
  uint64_t * d_lastQ, * d_nextQ;
  hipMalloc(&d_sums, maxblocks * sizeof(uint64_t));
  hipMalloc(&d_lastQ, num_p * sizeof(uint64_t));
  hipMalloc(&d_nextQ, num_p * sizeof(uint64_t));
  hipHostMalloc(&pq.h_primes, pq.len * sizeof(uint64_t));
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);
  hipStreamCreate(&stream[2]);

  hipMemcpy(pq.h_primes, pq.d_primes, pq.len*sizeof(uint64_t), hipMemcpyDeviceToHost);
  global::setXPlusB<<<num_p/threadsPerBlock + 1, threadsPerBlock, 0, stream[1]>>>(d_lastQ, num_p+1, (uint64_t)1);
  global::zero<<<maxblocks, threadsPerBlock, 0, stream[2]>>>(d_sums, maxblocks);

  hipDeviceSynchronize();
  KernelTime timer;
  timer.start();

  PiTable pi_table(sqrtx, pq.top);
  pi_table.set_pi_base(pi_sqrtx);

  // we need to get the maximum number of q values in a given iteration in order
  // to calculate the number of blocks needed.  abs((n/p)-(m/p)) is greatest at lowest
  // values of p, which obviously occur at the defined lower bound of p (p > x^(1/4)).
  // This is the value of p where we will define delta q so we don't miss any q's
  // by underestimating the number of blocks we need
  uint64_t pMax = sqrt(x/pi_table.getNextBaseDown());
  uint32_t pMaxIdx = (uint32_t)upperBound(pq.h_primes, 0, num_p, pMax);
  uint64_t qMax = mul128(sqrtx, qrtx) / pi_table.getNextBaseDown();
  uint32_t qMaxIdx = (uint32_t)upperBound(pq.h_primes, 0, pq.len, qMax);
  uint64_t qMinIdx = 0;

  // find all (p,q) pairs such that x/(p * q) >= x^(3/8)
  while(pi_table.get_base() > pq.top){
    // nextQ is copied from lastQ each iteration rather than switching pointers
    // to form the basis of a compare and swap operation in the kernel that evaluates
    // whether a given value has changed
    hipMemcpyAsync(d_nextQ, d_lastQ, num_p * sizeof(uint64_t), hipMemcpyDeviceToDevice, stream[1]);

    // get this iteration's pi table and bounds
    uint64_t pi_max = pi_table.get_pi_base();
    uint32_t * d_piTable = pi_table.getNextDown();

    // calculate number of blocks to span maximum range of q values (defined at
    // p = x^(1/4))
    uint32_t blocks = (qMaxIdx - qMinIdx)/threadsPerBlock + 1;
    std::cout << pi_table.get_base() << " " << pi_table.get_bottom() << " " << blocks << " " << qMaxIdx << " " << pMaxIdx << std::endl;

    // launch kernel
    hipDeviceSynchronize();
    hipProfilerStart();
    A_large_loPQ<<<blocks, threadsPerBlock, 0, stream[0]>>>
      (x, y, pq.d_primes, d_piTable, (pi_table.get_pi_base() & (~1ull)), pi_table.get_base(),
      pMaxIdx, d_sums, d_nextQ, d_lastQ, (uint64_t)pq.len);
    hipProfilerStop();
    // calculate the minimum and maximum p values and indices for next iteration
    qMinIdx = qMaxIdx;
    pMax = sqrt(x/pi_table.getNextBaseDown());
    pMaxIdx = upperBound(pq.h_primes, 0, num_p, pMax);
    qMax = mul128(sqrtx, qrtx) / pi_table.getNextBaseDown();
    qMaxIdx = upperBound(pq.h_primes, 0, pq.len, qMax);
    hipDeviceSynchronize();

  }

  sum = thrust::reduce(thrust::device, d_sums, d_sums + maxblocks);
  std::cout << "Low PQ:\t" << sum << std::endl;

  //
  global::zero<<<maxblocks, threadsPerBlock, 0, stream[0]>>>(d_sums, maxblocks);
  //

  pi_table.set_bottom(cbrtx);
  pMax = sqrt(x/pi_table.getNextBaseDown());
  pMaxIdx = upperBound(pq.h_primes, 0, num_p, pMax);
  uint64_t pMin = 0;
  uint32_t pMinIdx = 0;

  while(pi_table.get_base() > cbrtx){
    hipMemcpyAsync(d_nextQ, d_lastQ, num_p * sizeof(uint64_t), hipMemcpyDeviceToDevice, stream[1]);

    // get this iteration's pi table and bounds
    uint64_t pi_max = pi_table.get_pi_base();
    uint32_t * d_piTable = pi_table.getNextDown();

    // calculate number of blocks to span maximum range of q values (defined at
    // p = x^(1/4))
    uint32_t blocks = (pMaxIdx - pMinIdx) / threadsPerBlock + 1;
    std::cout << "Blocks : " << blocks << std::endl;

    // launch kernel
    hipDeviceSynchronize();
    hipProfilerStart();
    A_large_hiPQ_vert<<<blocks, threadsPerBlock, 0, stream[0]>>>
      (x, y, pq.d_primes, d_piTable, pi_table.get_pi_base(), pi_table.get_base(),
      pMinIdx, pMaxIdx, d_sums, d_nextQ, d_lastQ, (uint32_t)pq.len);
    hipProfilerStop();
    // calculate the minimum and maximum p values and indices for next iteration

    pMax = sqrt(x/pi_table.getNextBaseDown());
    pMaxIdx = (uint32_t)upperBound(pq.h_primes, 0, num_p, pMax);
    pMin = x / (pi_table.getNextBaseDown() * pi_table.getNextBaseDown());
    pMinIdx = upperBound(pq.h_primes, 0, num_p, pMin);
    hipDeviceSynchronize();
  }

  uint128_t sum2 = thrust::reduce(thrust::device, d_sums, d_sums + maxblocks);
  std::cout << "Hi PQ:\t" << sum2 << std::endl;

  timer.stop();
  timer.displayTime();
  timer.~KernelTime();
  pi_table.~PiTable();

  hipFree(d_sums);
  hipFree(pq.d_primes);
  hipHostFree(pq.h_primes);

  hipDeviceReset();

  return sum + sum2;
}

__global__
void A_large_loPQ(uint128_t x, uint64_t y, uint64_t * pq, uint32_t * d_pitable,
                  uint64_t pi_0, uint64_t base, uint32_t pMaxIdx, uint64_t * sums,
                  uint64_t * nextQ, uint64_t * lastQ, uint64_t maxQidx)
{
  uint64_t sum = 0;
  uint32_t tidx = threadIdx.x + blockDim.x * blockIdx.x;
  __shared__ uint64_t s_pi_chi[numThreads];
  s_pi_chi[threadIdx.x] = 0;
  __shared__ uint64_t s_lastQ[numThreads];

  for(uint64_t j = 0; j < pMaxIdx - 1; j += numThreads){
    s_lastQ[threadIdx.x] = (uint64_t)-1;
    __syncthreads();
    for(uint64_t i = j; i < min((uint32_t)pMaxIdx, (uint32_t)j + numThreads); i++){

      uint64_t qidx = nextQ[i] + tidx;
      if(qidx >= maxQidx) {atomicCAS((unsigned long long *)&s_lastQ[0], (unsigned long long)-1, (unsigned long long)maxQidx); break;}
      uint64_t q = pq[qidx];
      uint64_t p = pq[i];

      // calculate x/(p * q) and store value in q
      q = div128to64(x, (p * q));

      // check to make sure quotient is > pi_0, and coordinate this block's value
      // of lastQ if not
      q = checkRange(q, base, s_lastQ[i % numThreads], qidx);

      // calculate pi(x/(p * q)) * chi(x/(p * q)) if q is in range
      if(q != 0)
        s_pi_chi[threadIdx.x] += 1;//calculatePiChi(q, y, d_pitable, pi_0, base);
    } // repeat for all p values in range
    __syncthreads();

    // get global minimum value of lastQ
    minLastQ(j, s_lastQ, nextQ, lastQ);
  }
  __syncthreads();
  sum = thrust::reduce(thrust::device, s_pi_chi, s_pi_chi + numThreads);
  if(threadIdx.x == 0)
    sums[blockIdx.x] += sum;
}

__global__
void A_large_hiPQ_vert( uint128_t x, uint64_t y, uint64_t * pq, uint32_t * d_pitable,
                        uint64_t pi_0, uint64_t base, uint32_t pMinIdx, uint32_t pMaxIdx,
                        uint64_t * sums, uint64_t * nextQ, uint64_t * lastQ, uint64_t maxQidx)
{
  uint32_t tidx = threadIdx.x + blockDim.x * blockIdx.x;
  __shared__ uint64_t s_pi_chi[numThreads];
  s_pi_chi[threadIdx.x] = 0;

  uint64_t p, qidx, maxQ;
  if(tidx + pMinIdx > pMaxIdx) goto hiPQ_end;
  p = pq[tidx + pMinIdx];
  qidx = max((unsigned long long)tidx + pMinIdx + 1, (unsigned long long)nextQ[tidx + pMinIdx]);
  maxQ = min(x / (p * base), _isqrt(x/p));
  // printf("%llu %llu\n", tidx, maxQ);
  while(qidx < maxQidx){
    uint64_t q = pq[qidx];
    if(q > maxQ) break;
    // calculate x/(p * q) and store value in q
    q = div128to64(x, (p * q));

    // calculate pi(x/(p * q)) * chi(x/(p * q)) if q is in range
    s_pi_chi[threadIdx.x] += calculatePiChi(q, y, d_pitable, pi_0, base);

    qidx++;
    q = pq[qidx];
  } // repeat for all q values in range

  lastQ[tidx] = qidx;
  __syncthreads();
hiPQ_end:
  // repurpose p as the sum for this block
  p = thrust::reduce(thrust::device, s_pi_chi, s_pi_chi + numThreads);
  if(threadIdx.x == 0)
    sums[blockIdx.x] += p;
}

__device__
inline uint64_t checkRange(uint64_t q, uint64_t base, uint64_t & s_lastQ, uint64_t qidx)
{
  if(q + 1 < base){
    atomicMin((unsigned long long *)&s_lastQ, (unsigned long long)qidx);
    q = 0;
  }
  return q;
}

__device__
inline uint64_t calculatePiChi(uint64_t q, uint64_t y, uint32_t * d_pitable,
                                uint64_t pi_0, uint64_t base)
{
  // uint64_t r = d_pitable[(q + 1 - (base & ~1ull))/2] + pi_0;

  // for some reason doing this with ptx cuts about 5% off overall run time
  uint64_t r;
  uint32_t *ptr = &d_pitable[(q + 1 - (base & ~1ull))/2];
  asm("ld.global.u32.ca   %0, [%1];\n\t"
       : "=l" (r)
       : "l" (ptr));
  r += pi_0;

  if(q < y)
    r <<= 1;
  return r;
}

__device__
inline void minLastQ(uint32_t j, uint64_t * s_lastQ, uint64_t * nextQ, uint64_t * lastQ)
{
  uint32_t i = j + threadIdx.x;
  if(s_lastQ[threadIdx.x] != ~0){
    atomicCAS((unsigned long long *)&lastQ[i], (unsigned long long)nextQ[i], (unsigned long long)s_lastQ[threadIdx.x]);
    atomicMin((unsigned long long *)&lastQ[i], (unsigned long long)s_lastQ[threadIdx.x]);
  }
}
